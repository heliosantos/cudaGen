$!DECLARE_TIMER!$
	hipEvent_t start, stop;
	float elapsedTime;
$!CREATE_TIMER!$
	/* create the timers */
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	/* start the timer */
	HANDLE_ERROR(hipEventRecord(start, 0));
$!TERMINATE_TIMER!$
	HANDLE_ERROR(hipEventRecord(stop, 0));
	hipEventSynchronize(stop);
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	printf(\"execution took %3.6f miliseconds\", elapsedTime);
$!FREE_TIMER!$
	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));
