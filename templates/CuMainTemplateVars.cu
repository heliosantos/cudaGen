$!MEASURE_DECLARE_TIMER!$
	hipEvent_t start, stop;
	float elapsedTime;
$!MEASURE_CREATE_TIMER!$
	/* create the timers */
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	/* start the timer */
	HANDLE_ERROR(hipEventRecord(start, 0));
$!MEASURE_TERMINATE_TIMER!$
	HANDLE_ERROR(hipEventRecord(stop, 0));
	hipEventSynchronize(stop);
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	printf(\"execution took %3.6f miliseconds\", elapsedTime);
$!MEASURE_FREE_TIMER!$
	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));
